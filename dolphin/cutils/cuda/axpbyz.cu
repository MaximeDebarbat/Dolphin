#include "hip/hip_runtime.h"

#include <stdint.h>

__global__ void axpbyz_{{ dtype }}({{ dtype }} *x,
                                   {{ dtype }} *y,
                                   {{ dtype }} *z,
                                   {{ dtype }} a,
                                   {{ dtype }} b,
                                   uint32_t n){

    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        z[i] = a*x[i] + b*y[i];
    }
}
