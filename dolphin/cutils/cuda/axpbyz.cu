#include "hip/hip_runtime.h"

#include <stdint.h>

__global__ void axpbyz_{{ dtype }}(const {{ dtype }}* __restrict__ x,
                                  const {{ dtype }}* __restrict__ y,
                                  {{ dtype }}* __restrict__ z,
                                  const {{ dtype }} a,
                                  const {{ dtype }} b,
                                  const uint32_t* __restrict__ shape_x,
                                  const uint32_t* __restrict__ strides_x,
                                  const uint32_t* __restrict__ shape_y,
                                  const uint32_t* __restrict__ strides_y,
                                  const uint32_t* __restrict__ shape_z,
                                  const uint32_t* __restrict__ strides_z,
                                  const uint32_t ndim,
                                  const uint32_t n){

    for(uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){

        uint32_t index_x = index_transform(i, strides_x, shape_x, ndim);
        uint32_t index_y = index_transform(i, strides_y, shape_y, ndim);
        uint32_t index_z = index_transform(i, strides_z, shape_z, ndim);

        z[index_z] = a*x[index_x] + b*y[index_y];
    }
}
